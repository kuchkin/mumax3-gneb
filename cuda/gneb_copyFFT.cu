
#include <hip/hip_runtime.h>
// 3D micromagnetic kernel multiplication:
//
// |Mx|   |Kxx Kxy Kxz|   |Mx|
// |My| = |Kxy Kyy Kyz| * |My|
// |Mz|   |Kxz Kyz Kzz|   |Mz|
//
// ~kernel has mirror symmetry along Y and Z-axis,
// apart form first row,
// and is only stored (roughly) half:
//
// K11, K22, K02:
// xxxxx
// aaaaa
// bbbbb
// ....
// bbbbb
// aaaaa
//
// K12:
// xxxxx
// aaaaa
// bbbbb
// ...
// -bbbb
// -aaaa

extern "C" __global__ void
copyFFT(float* __restrict__  fft2x,  float* __restrict__  fft2y,  float* __restrict__  fft2z,
                float* __restrict__  fftx,  float* __restrict__  ffty,  float* __restrict__  fftz,
               int noi, int image, int Nx, int Ny, int Nz) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if(ix>= Nx || iy>= Ny || iz>=Nz) {
        return;
    }
    if(iz/(Nz/noi) != image) return;

    // fetch (complex) FFT'ed magnetization
    int I = (iz*Ny + iy)*Nx + ix;
    int e = 1 * I;
    

    // m * K matrix multiplication, overwrite m with result.
    fftx[e  ] = fft2x[e  ] ;
    // fftx[e+1] = fft2x[e+1] ;
    ffty[e  ] = fft2y[e  ] ;
    // ffty[e+1] = fft2y[e+1] ;
    fftz[e  ] = fft2z[e  ] ;
    // fftz[e+1] = fft2z[e+1] ;
}

