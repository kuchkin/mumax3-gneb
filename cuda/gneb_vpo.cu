#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"



// Descent energy minimizer
extern "C" __global__ void
vpo(float* __restrict__ mx,  float* __restrict__  my,  float* __restrict__ mz,
         float* __restrict__ Bx,  float* __restrict__  By,  float* __restrict__ Bz,uint8_t* regions,
          float dt, int minend, int noi, int N, int Nz) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    // bool pp = true;
    
    if (i < N) {


        float3 m = {mx[i], my[i], mz[i]};
        float3 B = {Bx[i], By[i], Bz[i]};


        //simple
        // m += dt*B;
        // m = normalized(m);
        bool p;
        if(minend == 0 ){
            p = true;
            // p = ( (((i/(N/Nz))/(Nz/noi)) == 0 ) || (((i/(N/Nz))/(Nz/noi)) == (noi-1) ) )? false : true;
        }else{
            p = ( (((i/(N/Nz))/(Nz/noi)) == 0 ) || (((i/(N/Nz))/(Nz/noi)) == (noi-1) ) )? true : false;
        }

        //rotation rodrigues

        if(p){


        float theta = dt*len(B);
        float pref;
        if(theta<=0.005){//1e-5
            pref = 1.0 - theta*theta*(1.0-theta*theta/20.0)/6.0;
            
        }else{
            pref = sin(theta)/theta;
        }

        pref *= dt;

        
        m = m*cos(theta) + B*pref;
        if(regions[i] == 255){
            m.x = 0.0; m.y = 0.0; m.z = 1.0;
        }
        
        m = normalized(m);
        mx[i] = m.x;
        my[i] = m.y;
        mz[i] = m.z;
        }

        //rotation matrix
        // float theta = dt*len(B);

        // float q = cos(theta), w = 1-q;
        // float x = -dt*Bx[i]/theta, y = -dt*By[i]/theta, z = -dt*Bz[i]/theta;
        // float s1 = -y*z*w, s2 = x*z*w, s3 = -x*y*w;
        // float p1 = x*sin(theta), p2 = y*sin(theta), p3 = z*sin(theta);

        // float t1, t2, t3;
        // if(theta > 1.0e-20){
        //     t1 = (q+z*z*w) * mx[i] + (s1+p1)   * my[i] + (s2+p2)   * mz[i];
        //     t2 = (s1-p1)   * mx[i] + (q+y*y*w) * my[i] + (s3+p3)   * mz[i];
        //     t3 = (s2-p2)   * mx[i] + (s3-p3)   * my[i] + (q+x*x*w) * mz[i];
        //     mx[i] = t1;
        //     my[i] = t2;
        //     mz[i] = t3;
        // };

        //rotation
        // float theta = dt*len(B);
        // if(theta < 1e-20){
        //     m += dt*B;
        //     m = normalized(m);
        // }else{
        //     float3 vecnom = cross(m,B)*(1./len(B));
        //     m = m*cos(theta) + cross(vecnom,m)*sin(theta);
        // }

        // mx[i] = m.x;
        // my[i] = m.y;
        // mz[i] = m.z;
        


    }
}
