#include "hip/hip_runtime.h"

#include "float3.h"


// dst += prefactor * dot(a,b)
extern "C" __global__ void
getphi(float* __restrict__ dst,float* __restrict__ src, float prefactor,
           float* __restrict__ ax, float* __restrict__ ay, float* __restrict__ az,
           float* __restrict__ bx, float* __restrict__ by, float* __restrict__ bz,
           int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {
        float3 A = {ax[i], ay[i], az[i]};
        float3 B = {bx[i], by[i], bz[i]};

        float3 vecnom = cross(A, B);
        float SIN = sqrt(vecnom.x*vecnom.x + vecnom.y*vecnom.y + vecnom.z*vecnom.z)/(len(A)*len(B));

        float at = atan2f(SIN, src[i]);

        dst[i] = at*at;
    }
}

