#include "hip/hip_runtime.h"
#include <stdint.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"
#include "amul.h"

// See exchange.go for more details.
extern "C" __global__ void
gneb_addexchange(float* __restrict__ Bx, float* __restrict__ By, float* __restrict__ Bz,
            float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
            float* __restrict__ Ms_, float Ms_mul,
            float* __restrict__ aLUT2d, uint8_t* __restrict__ regions,
            float wx, float wy, float wz, int Nx, int Ny, int Nz,int noi, uint8_t PBC, uint8_t GNEB, float JZ) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }

    // central cell
    int I = idx(ix, iy, iz);
    float3 m0 = make_float3(mx[I], my[I], mz[I]);

    if (is0(m0)) {
        return;
    }

    uint8_t r0 = regions[I];
    float3 B  = make_float3(0.0,0.0,0.0);
    // float3 T  = make_float3(0.0,0.0,0.0);

    int i_;    // neighbor index
    float3 m_; // neighbor mag
    float a__; // inter-cell exchange stiffness

    float3 m1;
    float3 m2;

    // left neighbor
    i_  = idx(lclampx(ix-1), iy, iz);           // clamps or wraps index according to PBC
    m_  = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_  = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    a__ = aLUT2d[symidx(r0, regions[i_])];
    B += wx * a__ *(m_ - m0);

    // right neighbor
    i_  = idx(hclampx(ix+1), iy, iz);
    m_  = make_float3(mx[i_], my[i_], mz[i_]);
    m_  = ( is0(m_)? m0: m_ );
    a__ = aLUT2d[symidx(r0, regions[i_])];
    B += wx * a__ *(m_ - m0);

    // back neighbor
    i_  = idx(ix, lclampy(iy-1), iz);
    m_  = make_float3(mx[i_], my[i_], mz[i_]);
    m_  = ( is0(m_)? m0: m_ );
    a__ = aLUT2d[symidx(r0, regions[i_])];
    B += wy * a__ *(m_ - m0);

    // front neighbor
    i_  = idx(ix, hclampy(iy+1), iz);
    m_  = make_float3(mx[i_], my[i_], mz[i_]);
    m_  = ( is0(m_)? m0: m_ );
    a__ = aLUT2d[symidx(r0, regions[i_])];
    B += wy * a__ *(m_ - m0);

    // only take vertical derivative for 3D sim
    if (Nz != 1 && !GNEB2D && !GNEB3D) {
        // bottom neighbor
        i_  = idx(ix, iy, lclampz(iz-1));
        m_  = make_float3(mx[i_], my[i_], mz[i_]);
        m_  = ( is0(m_)? m0: m_ );
        a__ = aLUT2d[symidx(r0, regions[i_])];
        B += wz * a__ *(m_ - m0);

        // top neighbor
        i_  = idx(ix, iy, hclampz(iz+1));
        m_  = make_float3(mx[i_], my[i_], mz[i_]);
        m_  = ( is0(m_)? m0: m_ );
        a__ = aLUT2d[symidx(r0, regions[i_])];
        B += wz * a__ *(m_ - m0);
    }
    //&& Nz%noi == 0
    if (Nz != 1 && GNEB3D ) {
        // bottom neighbor
        // i_  = idx(ix, iy, max(iz%(Nz/noi)-1,0) + (iz/(Nz/noi))*Nz/noi);
        i_  = idx(ix, iy, lclamp(iz%(Nz/noi)-1,Nz/noi) + (iz/(Nz/noi))*Nz/noi);
        m_  = make_float3(mx[i_], my[i_], mz[i_]);
        m_  = ( is0(m_)? m0: m_ );
        a__ = aLUT2d[symidx(r0, regions[i_])];
        B += JZ*wz * a__ *(m_ - m0);

        // top neighbor
        // i_  = idx(ix, iy, min(iz%(Nz/noi)+1,Nz/noi-1) + (iz/(Nz/noi))*Nz/noi);
        i_  = idx(ix, iy, hclamp(iz%(Nz/noi)+1,Nz/noi) + (iz/(Nz/noi))*Nz/noi);
        m_  = make_float3(mx[i_], my[i_], mz[i_]);
        m_  = ( is0(m_)? m0: m_ );
        a__ = aLUT2d[symidx(r0, regions[i_])];
        B += JZ*wz * a__ *(m_ - m0);
    }

    // if(GNEB2D && iz !=0 && iz != (Nz-1)){
    //     i_  = idx(ix, iy, iz-1);
    //     m1 = make_float3(mx[i_], my[i_], mz[i_]);
    //     i_  = idx(ix, iy, iz+1);
    //     m2 = make_float3(mx[i_], my[i_], mz[i_]);
    //     T = normalized((m2-m1) - dot((m2-m1), m0)*m0);
    // }

    float invMs = inv_Msat(Ms_, Ms_mul, I);
    float par = 1.0;
    // if(GNEB2D || GNEB3D) par = 0.5;
    Bx[I] += B.x*invMs*par;
    By[I] += B.y*invMs*par;
    Bz[I] += B.z*invMs*par;
}

