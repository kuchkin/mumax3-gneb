#include "hip/hip_runtime.h"
#include "float3.h"

// normalize vector {vx, vy, vz} to unit length, unless length or vol are zero.
extern "C" __global__ void
random4D(float* __restrict__ vx, float* __restrict__ vy, float* __restrict__ vz, float* __restrict__ vol,float* __restrict__ v4, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {

        // float v = (vol == NULL? 1.0f: vol[i]);
        // float3 V = {v*vx[i], v*vy[i], v*vz[i]};
        // float norm = v/sqrt(V.x*V.x+V.y*V.y+V.z*V.z+v4[i]*v4[i]);
        // vx[i] *= norm;
        // vy[i] *= norm;
        // vz[i] *= norm;
        v4[i] = vx[i];
    }
}

