#include "hip/hip_runtime.h"
#include <stdint.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"
#include "amul.h"

#define MAX(i, j) ( (i>j)? (i)  :  (j) )
#define MIN(i, j) ( (i<j)? (i)  :  (j) )
#define ABS(a) ( (a>0.0)? (a)  :  (-1.0*a) )

// See gneb.go for more details.




extern "C" __global__ void
gneb(float* __restrict__ Bx, float* __restrict__ By, float* __restrict__ Bz,
    float* __restrict__ Tx, float* __restrict__ Ty, float* __restrict__ Tz,
            float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
            int Nx, int Ny, int Nz,int noi,int image, float Tp,
            float Lp,float Ln,float k,int CIGNEB, int Pos){

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;
    

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }
    if(iz/(Nz/noi) != image){
        return;
    }

 
    int I = idx(ix, iy, iz);

    float3 T  = make_float3(Tx[I]/sqrtf(Tp),Ty[I]/sqrtf(Tp),Tz[I]/sqrtf(Tp));
    float3 m0 = make_float3(mx[I], my[I], mz[I]);
    float3 B  = make_float3(Bx[I], By[I], Bz[I]);
    
    float pref;
    if(CIGNEB == 1 && iz/(Nz/noi) == Pos){
        pref = -2.*dot(B,T);
    }else{
        pref = k*(Ln-Lp) + dot(B,T);
    }
    
    B.x = B.x + pref*T.x;
    B.y = B.y + pref*T.y;
    B.z = B.z + pref*T.z;
    
    
    
    // The perpendicular component of the energy gradient
    // B = B - dot(B,T)*T;
    // T = k*(sqrtf(Lp)-sqrtf(Ln))*T + B;
    // T = k*(Lp-Ln)*T + B;

    Bx[I] = B.x - dot(m0,B)*m0.x;
    By[I] = B.y - dot(m0,B)*m0.y;
    Bz[I] = B.z - dot(m0,B)*m0.z;

    
}


