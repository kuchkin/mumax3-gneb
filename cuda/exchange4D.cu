#include "hip/hip_runtime.h"
#include <stdint.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"
#include "amul.h"

// See exchange.go for more details.
extern "C" __global__ void
addexchange4D(float* __restrict__ n, float* __restrict__ Bx,
            float* __restrict__ Ms_, float Ms_mul, float kappa,
            float* __restrict__ aLUT2d, uint8_t* __restrict__ regions,
            float wx, float wy, float wz, int Nx, int Ny, int Nz, uint8_t PBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }

    // central cell
    int I = idx(ix, iy, iz);
    float3 m0 = make_float3(n[I], 0.0, 0.0);
    // if (is0(m0)) {
    //     return;
    // }

    uint8_t r0 = regions[I];
    float3 B  = make_float3(0.0,0.0,0.0);

    int i_;    // neighbor index
    float3 m_; // neighbor mag
    float a__; // inter-cell exchange stiffness

    // left neighbor
    i_  = idx(lclampx(ix-1), iy, iz);           // clamps or wraps index according to PBC
    m_  = make_float3(n[i_], 0.0, 0.0);  // load m
    m_  = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    a__ = aLUT2d[symidx(r0, regions[i_])];
    B += wx * a__ *(m_ - m0);

    // right neighbor
    i_  = idx(hclampx(ix+1), iy, iz);
    m_  = make_float3(n[i_], 0.0, 0.0);
    m_  = ( is0(m_)? m0: m_ );
    a__ = aLUT2d[symidx(r0, regions[i_])];
    B += wx * a__ *(m_ - m0);

    // back neighbor
    i_  = idx(ix, lclampy(iy-1), iz);
    m_  = make_float3(n[i_], 0.0, 0.0);
    m_  = ( is0(m_)? m0: m_ );
    a__ = aLUT2d[symidx(r0, regions[i_])];
    B += wy * a__ *(m_ - m0);

    // front neighbor
    i_  = idx(ix, hclampy(iy+1), iz);
    m_  = make_float3(n[i_], 0.0, 0.0);
    m_  = ( is0(m_)? m0: m_ );
    a__ = aLUT2d[symidx(r0, regions[i_])];
    B += wy * a__ *(m_ - m0);

    // only take vertical derivative for 3D sim
    if (Nz != 1) {
        // bottom neighbor
        i_  = idx(ix, iy, lclampz(iz-1));
        m_  = make_float3(n[i_], 0.0, 0.0);
        m_  = ( is0(m_)? m0: m_ );
        a__ = aLUT2d[symidx(r0, regions[i_])];
        B += wz * a__ *(m_ - m0);

        // top neighbor
        i_  = idx(ix, iy, hclampz(iz+1));
        m_  = make_float3(n[i_], 0.0, 0.0);
        m_  = ( is0(m_)? m0: m_ );
        a__ = aLUT2d[symidx(r0, regions[i_])];
        B += wz * a__ *(m_ - m0);
    }

    float invMs = inv_Msat(Ms_, Ms_mul, I);
    Bx[I] = B.x*invMs;
    Bx[I] -= 2.0*kappa*n[I]*invMs;
    return;
}

