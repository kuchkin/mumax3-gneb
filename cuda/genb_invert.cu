#include "hip/hip_runtime.h"

#include "float3.h"

// dst += prefactor * dot(a,b)
extern "C" __global__ void
invert(float* __restrict__ dst, float* __restrict__ src, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {
        if(src[i]==0.) {
            dst[i] = 0.0;
        }else{
            dst[i] = 1./src[i];
        }
    }
}

