#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"



// Descent energy minimizer
extern "C" __global__ void
vpo4D(float* __restrict__ mx,  float* __restrict__  my,  float* __restrict__ mz, float* __restrict__ mw,
         float* __restrict__ Bx,  float* __restrict__  By,  float* __restrict__ Bz, float* __restrict__  Bw,
         uint8_t* regions, float dt,float vf, int minend, int noi, int N, int Nz) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    // bool pp = true;
    
    if (i < N) {

        // if((((i/(N/Nz))/(Nz/noi)) == 0 ) || (((i/(N/Nz))/(Nz/noi)) == (noi-1) )){

        //     float nx = mx[i], ny = my[i], nz = mz[i], nv = mw[i];
        //     float tx = Bx[i], ty = By[i], tz = Bz[i], tv = Bw[i];

        //     float d1,d2,d3;
            
        //     float ss  = (nv > 0.0)? 1.0 : 1.0;
        //     float den = (nv > 0.0)? 1.0/(1.0 + nv) : 1.0/(1.0 - nv);
            
        //     float g1 = nx*den, g2 = ny*den, g3 = nz*den; 

        //     d1 = (-tx*(1.0+ss*nv - nx*nx) + ty*nx*ny + tz*nx*nz + tv*nx*(ss+nv));
        //     d2 = (tx*nx*ny - ty*(1.0+ss*nv - ny*ny) + tz*ny*nz + tv*ny*(ss+nv));
        //     d3 = (tx*nx*nz + ty*ny*nz - tz*(1.0+ss*nv - nz*nz) + tv*nz*(ss+nv));

        //     g1 -= dt*d1; g2 -= dt*d2; g3 -= dt*d3;
            
        //     float gg = g1*g1+g2*g2+g3*g3;
        //     float gi = 1./(1.+gg);
        //     nx = 2.*g1*gi; ny = 2.*g2*gi; nz = 2.*g3*gi; nv = ss*(1.-gg)*gi;
        //     mx[i] = nx;
        //     my[i] = ny;
        //     mz[i] = nz;
        //     mw[i] = nv;

        // }else if(minend == 0){
        //     Bx[i] = Bx[i]*vf; By[i] = By[i]*vf; Bz[i] = Bz[i]*vf; Bw[i] = Bw[i]*vf;

        //     float3 m = {mx[i], my[i], mz[i]};
        //     float3 B = {Bx[i], By[i], Bz[i]};


        //     float theta = dt*len4D(B, Bw[i]);
        //     float pref;
        //     if(theta<=0.005){//1e-5
        //         pref = 1.0 - theta*theta*(1.0-theta*theta/20.0)/6.0;
                
        //     }else{
        //         pref = sin(theta)/theta;
        //     }

        //     pref *= dt;
            
        //     m = m*cos(theta) + B*pref;
        //     float m4 = mw[i]*cos(theta) + Bw[i]*pref;

        //     if(regions[i] == 255){
        //         m.x = 0.0; m.y = 0.0; m.z = 1.0; m4 = 0.0;
        //     }
        //     float norm = 1.0/len4D(m, m4);
        //     // m = normalized(m);
        //         mx[i] = m.x*norm;
        //         my[i] = m.y*norm;
        //         mz[i] = m.z*norm;
        //         mw[i] =  m4*norm;
        //     }

        Bx[i] = Bx[i]*vf; By[i] = By[i]*vf; Bz[i] = Bz[i]*vf; Bw[i] = Bw[i]*vf;

            float3 m = {mx[i], my[i], mz[i]};
            float3 B = {Bx[i], By[i], Bz[i]};


            float theta = dt*len4D(B, Bw[i]);
            float pref;
            if(theta<=0.005){//1e-5
                pref = 1.0 - theta*theta*(1.0-theta*theta/20.0)/6.0;
                
            }else{
                pref = sin(theta)/theta;
            }

            pref *= dt;
            
            m = m*cos(theta) + B*pref;
            float m4 = mw[i]*cos(theta) + Bw[i]*pref;

            if(regions[i] == 255){
                m.x = 0.0; m.y = 0.0; m.z = 1.0; m4 = 0.0;
            }
            float norm = 1.0/len4D(m, m4);
            // m = normalized(m);
                mx[i] = m.x*norm;
                my[i] = m.y*norm;
                mz[i] = m.z*norm;
                mw[i] =  m4*norm;
        }

}
