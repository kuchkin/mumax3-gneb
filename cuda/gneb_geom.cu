#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"



// Descent energy minimizer
extern "C" __global__ void
geom_vpo(float* __restrict__ Bx,  float* __restrict__  By,  float* __restrict__ Bz,float* __restrict__ vol,
          int N, int Nz) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    // bool pp = true;
    
    if (i < N) {


 
        Bx[i] *= vol[i];
        By[i] *= vol[i];
        Bz[i] *= vol[i];
        


    }
}
