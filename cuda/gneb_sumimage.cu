#include "hip/hip_runtime.h"
#include "reduceim2.h"
#include "sum.h"

#define load(i) src[i]
// #define loadim(i,image, noi,Nz,n) ((((i/(n/Nz))/(Nz/noi)) == image)? (src[i]) : (0.0))


extern "C" __global__ void
sumimage(float* __restrict__ src, float*__restrict__  dst, float initVal, int n, int image, int noi, int Nz) {
    reduceim2(load, sum, atomicAdd,image,noi,Nz)
}

