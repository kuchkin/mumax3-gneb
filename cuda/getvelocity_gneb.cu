#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"



// velocity rotate
extern "C" __global__ void
get_velocity(float* __restrict__ vx,  float* __restrict__  vy,  float* __restrict__ vz,
    float* __restrict__ kx,  float* __restrict__  ky,  float* __restrict__ kz,
         float* __restrict__ m1,  float* __restrict__  m2,
         int N, int Nz) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    
    if (i < N) {
        float vf = 1.0;
        if(m1[i]<0){
            vf = 0.0;
        }else{
            vf = m1[i]/m2[i];
        }
        vx[i] = vf*kx[i];
        vy[i] = vf*ky[i];
        vz[i] = vf*kz[i];


    }
}
