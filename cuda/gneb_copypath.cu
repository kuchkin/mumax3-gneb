#include "hip/hip_runtime.h"
#include <stdint.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"
#include "amul.h"

#define MAX(a, b) ( (a>b)? (a)  :  (b) )
#define MIN(a, b) ( (a<b)? (a)  :  (b) )
#define ABS(a) ( (a>0.0)? (a)  :  (-1.0*a) )

// See gneb.go for more details.




extern "C" __global__ void
copypath(float* __restrict__ Tx, float* __restrict__ Ty, float* __restrict__ Tz,
            float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
            int Nx, int Ny, int Nz,int noi){

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;
    

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }
    int I = idx(ix, iy, iz);
    if(iz/(Nz/noi) == 0 || iz/(Nz/noi) == (noi-1)){
        Tx[I] = 0.0;
        Ty[I] = 0.0;
        Tz[I] = 0.0;
    }else{
        Tx[I] = mx[I];
        Ty[I] = my[I];
        Tz[I] = mz[I];
    }

    
}


