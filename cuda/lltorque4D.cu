#include "hip/hip_runtime.h"
#include "amul.h"
#include "float3.h"
#include <stdint.h>

// Landau-Lifshitz torque.
extern "C" __global__ void
lltorque4D(float* __restrict__ n, float* __restrict__ l, 
           float* __restrict__  tx, float* __restrict__  ty, float* __restrict__  tz,
          float* __restrict__  mx, float* __restrict__  my, float* __restrict__  mz,
          float* __restrict__  hx, float* __restrict__  hy, float* __restrict__  hz,
          float* __restrict__  alpha_, float alpha_mul, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {

        float3 m = make_float3(mx[i], my[i], mz[i]);
        float3 H = make_float3(hx[i], hy[i], hz[i]);
        float alpha = amul(alpha_, alpha_mul, i);

        float3 mxH = cross(m, H);
        float gilb = -1.0f / (1.0f + alpha * alpha);

        float mB = mx[i]*hx[i] + my[i]*hy[i] + mz[i]*hz[i] + n[i]*l[i];

        // float3 torque = gilb * (mxH + alpha * cross(m, mxH));

        tx[i] = gilb*(mxH.x + alpha*(mB*m.x - H.x));
        ty[i] = gilb*(mxH.y + alpha*(mB*m.y - H.y));
        tz[i] = gilb*(mxH.z + alpha*(mB*m.z - H.z));
        l[i]  = gilb*alpha*(mB*n[i] - l[i]);
    }
}

