#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"



// generate u1 u2
extern "C" __global__ void
rotate(float* __restrict__ sx,  float* __restrict__  sy,  float* __restrict__ sz,
         float* __restrict__ vx,  float* __restrict__  vy,  float* __restrict__ vz,
         float* __restrict__ sxt,  float* __restrict__  syt,  float* __restrict__ szt,
         float* __restrict__ wx,  float* __restrict__  wy,  float* __restrict__ wz,
         float epsilon, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    // bool pp = true;
    
    if (i < N) {


        float kx = sy[i]*wz[i] - sz[i]*wy[i];
        float ky = sz[i]*wx[i] - sx[i]*wz[i];
        float kz = sx[i]*wy[i] - sy[i]*wx[i];

        float norm = 1.0/sqrt(kx*kx+ky*ky+kz*kz);
        kx *= norm; ky *= norm; kz *= norm;

        float bx = ky*vz[i]-vy[i]*kz;
        float by = kz*vx[i]-vz[i]*kx;
        float bz = kx*vy[i]-vx[i]*ky;
        float ww = sqrt(wx[i]*wx[i]+wy[i]*wy[i]+wz[i]*wz[i]);
        float COS = cos(epsilon*ww);
        float SIN = sin(epsilon*ww);
        float ks = kx*vx[i]+ky*vy[i]+kz*vz[i];

        sxt[i] = vx[i]*COS - bx*SIN + kx*ks*(1.-COS);
        syt[i] = vy[i]*COS - by*SIN + ky*ks*(1.-COS);
        szt[i] = vz[i]*COS - bz*SIN + kz*ks*(1.-COS);
    }
}
