#include "hip/hip_runtime.h"
#include "float3.h"

// normalize vector {vx, vy, vz} to unit length, unless length or vol are zero.
extern "C" __global__ void
sqrt4D(float* __restrict__ vx, float* __restrict__ vy, float* __restrict__ vz, float* __restrict__ vol,float* __restrict__ v4, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {

        // float v = (vol == NULL? 1.0f: vol[i]);
        // float3 V = {v*vx[i], v*vy[i], v*vz[i]};
        // float norm = v/sqrt(V.x*V.x+V.y*V.y+V.z*V.z+v4[i]*v4[i]);
        // vx[i] *= norm;
        // vy[i] *= norm;
        // vz[i] *= norm;
        float temp = vx[i]*vx[i]+vy[i]*vy[i]+vz[i]*vz[i];
        if (temp >= 1.0){
            v4[i] = 0.0;
        }else{
             v4[i] = sqrt(1.0-temp);
        }
       
    }
}

