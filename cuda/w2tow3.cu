#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"



// generate u1 u2
extern "C" __global__ void
w2tow3(float* __restrict__ u1x,  float* __restrict__  u1y,  float* __restrict__ u1z,
         float* __restrict__ u2x,  float* __restrict__  u2y,  float* __restrict__ u2z,
         float* __restrict__ w2x,  float* __restrict__  w2y,
         float* __restrict__ w3x,  float* __restrict__  w3y,  float* __restrict__ w3z,  int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    // bool pp = true;
    
    if (i < N) {

        w3x[i] = w2x[i]*u1x[i] + w2y[i]*u2x[i];
        w3y[i] = w2x[i]*u1y[i] + w2y[i]*u2y[i];
        w3z[i] = w2x[i]*u1z[i] + w2y[i]*u2z[i];


    }
}
