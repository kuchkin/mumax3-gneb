#include "hip/hip_runtime.h"
#include "reduceforce.h"
#include "sum.h"

#define load(i) src[i]

extern "C" __global__ void
totalforce(float* __restrict__ src, float*__restrict__  dst, float initVal, int n, int noi,int Nz) {
    reduceforce(load, sum, atomicAdd,noi)
}