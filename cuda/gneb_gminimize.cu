#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"



// Descent energy minimizer
extern "C" __global__ void
gminimize(float* __restrict__ mx,  float* __restrict__  my,  float* __restrict__ mz,
         float* __restrict__ Bx,  float* __restrict__  By,  float* __restrict__ Bz,
         float dt, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    bool pp = true;
    
    if (i < N) {

        // if((((i/(N/Nz))/(Nz/noi)) == 0 ) && MinimizeFirst == 0) pp = false;
        // if((((i/(N/Nz))/(Nz/noi)) == (noi-1) ) && MinimizeLast == 0) pp = false;
        // if(MinimizeEndPoints == 1){
        //     pp = false;
        //     if((((i/(N/Nz))/(Nz/noi)) == 0 ) && MinimizeFirst == 1) pp = true;
        //     if((((i/(N/Nz))/(Nz/noi)) == (noi-1) ) && MinimizeLast == 1) pp = true;
        // }
        // if(pp){
            float3 m = {mx[i], my[i], mz[i]};
            float3 B = {Bx[i], By[i], Bz[i]};

            float ss = 1.0;
            if(m.z<0) ss = -1.0;


            
            float g1 = m.x/(1.0 + ss*m.z);
            float g2 = m.y/(1.0 + ss*m.z);
            float d1 = (B.x*(m.y*m.y + ss*m.z*(1.0 + ss*m.z)) - B.y*m.x*m.y - B.z*m.x*(ss + m.z));
            float d2 = (-B.x*m.x*m.y + B.y*(m.x*m.x + ss*m.z*(1.0 + ss*m.z)) - B.z*m.y*(ss + m.z));
            g1 = g1 + dt*d1;
            g2 = g2 + dt*d2;
            d1 = 1./(1.+g1*g1+g2*g2);
            mx[i] = 2.*g1*d1;
            my[i] = 2.*g2*d1;
            mz[i] = ss*(1.-g1*g1-g2*g2)*d1;
        // }
        

        // m += dt*B;
        // m = normalized(m);
        // mx[i] = m.x;
        // my[i] = m.y;
        // mz[i] = m.z;


    }
}
