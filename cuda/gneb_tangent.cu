#include "hip/hip_runtime.h"
#include <stdint.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"
#include "amul.h"

#define MAX(a, b) ( (a>b)? (a)  :  (b) )
#define MIN(a, b) ( (a<b)? (a)  :  (b) )
#define ABS(a) ( (a>0.0)? (a)  :  (-1.0*a) )

// See gneb.go for more details.




extern "C" __global__ void
tangent(float* __restrict__ Tx, float* __restrict__ Ty, float* __restrict__ Tz,
            float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
            int Nx, int Ny, int Nz,int noi,int image, float Ep, float Ei, float En,
            float Lp,float Ln){

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;
    

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }
    if(iz/(Nz/noi) != image){
        return;
    }

 

    float3 Tp  = make_float3(0.0,0.0,0.0);
    float3 Tm  = make_float3(0.0,0.0,0.0);
    float3 T   = make_float3(0.0,0.0,0.0);

    int I = idx(ix, iy, iz);
    float3 m0 = make_float3(mx[I], my[I], mz[I]);
    int i_  = idx(ix, iy, iz-Nz/noi);
    float3 m1 = make_float3(mx[i_], my[i_], mz[i_]);
    i_  = idx(ix, iy, iz+Nz/noi);
    float3 m2 = make_float3(mx[i_], my[i_], mz[i_]);

    Tp = (m2-m0);
    Tm = (m0-m1);
    // Tp.x /= (Ln);
    // Tp.y /= (Ln);
    // Tp.z /= (Ln);
    // Tm.x /= (Lp);
    // Tm.y /= (Lp);
    // Tm.z /= (Lp);
    if(En>Ei && Ei>Ep){
        T = Tp;
    }else if(En<Ei && Ei<Ep){
        T = Tm;
    }else{
        float dEmax = MAX(ABS(En-Ei),ABS(Ep-Ei))+1e-8;
        float dEmin = MIN(ABS(En-Ei),ABS(Ep-Ei));
        // float pref  = 1.0;
        // if(sqrt(dEmax*dEmax+dEmin*dEmin)>0) pref = 1./sqrt(dEmax*dEmax+dEmin*dEmin);
        if(En>Ep){
            // T = (dEmax*Tp + dEmin*Tm)*pref;
            T = Tp + (dEmin/dEmax)*Tm;
            // T = Tp + (dEmin)*Tm;
        }else{
            //T = (dEmin*Tp + dEmax*Tm)*pref;
            T = (dEmin/dEmax)*Tp + Tm;
            // T = (dEmin)*Tp + Tm;
        }
    }

    float m0T = dot(m0,T);

    Tx[I] = T.x - m0T*m0.x;
    Ty[I] = T.y - m0T*m0.y;
    Tz[I] = T.z - m0T*m0.z;

    
}


