#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"



// generate u1 u2
extern "C" __global__ void
generate_w(float* __restrict__ w2x,  float* __restrict__  w2y,  int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    // bool pp = true;
    
    if (i < N) {

        w2x[i] = 1.0/sqrt(2.0*N);
        w2y[i] = 1.0/sqrt(2.0*N);


    }
}
