#include "hip/hip_runtime.h"
#include "stencil.h"

// shift dst by shy cells (positive or negative) along Z-axis.
// new edge value is clampL at left edge or clampR at right edge.
extern "C" __global__ void
shiftmagz4(float* __restrict__  dstX, float* __restrict__  srcX,
          int Nx,  int Ny,  int Nz, int shz, float clampL, float clampR) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if(ix < Nx && iy < Ny && iz < Nz) {
        int iz2 = iz-shz;
        float newval;
        if (iz2 < 0) {
            newval = clampL;
        } else if (iz2 >= Nz) {
            newval = clampR;

        } else {
            newval = srcX[idx(ix, iy, iz2)];
        }
        dstX[idx(ix, iy, iz)] = newval;
    }
}

