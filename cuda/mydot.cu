#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"



// Descent energy minimizer
extern "C" __global__ void
mydot(float* __restrict__ sm,float* __restrict__ ax,  float* __restrict__  ay,  float* __restrict__ az,
         float* __restrict__ bx,  float* __restrict__  by,  float* __restrict__ bz, int N, int Nz) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    // bool pp = true;
    float temp = 0.0, vv = 0.0;
    if (i == 0) {
        for(int j=0; j<N; j++){
            vv = ax[j]*bx[j] +ay[j]*by[j]+az[j]*bz[j];
            // if(vv!=1){
                temp += vv;
            // }
            
        }
        // if(temp>4095) temp = 100000.0;
        sm[0] = temp;
    }
}
