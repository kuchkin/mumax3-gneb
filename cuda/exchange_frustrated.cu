#include "hip/hip_runtime.h"
#include <stdint.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"
#include "amul.h"

// See exchange.go for more details.
extern "C" __global__ void
addexchange_frustrated(float* __restrict__ Bx, float* __restrict__ By, float* __restrict__ Bz,
            float* __restrict__ Sx, float* __restrict__ Sy, float* __restrict__ Sz,
            float* __restrict__ Ms_, float Ms_mul,
            float* __restrict__ aLUT2d, uint8_t* __restrict__ regions,
            float wx, float wy, float wz, int Nx, int Ny, int Nz, uint8_t PBC, 
            float J1, float J2, float J3, float J4) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= Nx || j >= Ny || k >= Nz) {
        return;
    }

    int I = idx(i, j, k), np, nm;
    int nnm1, nnm2, nnp1, nnp2;
    float sx = Sx[I], sy = Sy[I], sz = Sz[I];
    float bx = 0.0, by = 0.0, bz = 0.0;
    


    //Shell 1 (6 neighbours)
    if(Nx>1){
        nm = indexPBC(i-1, j, k, Nx, Ny, Nz);
        np = indexPBC(i+1, j, k, Nx, Ny, Nz);

        bx += Sx[nm] + Sx[np] - 2.0*sx;
        by += Sy[nm] + Sy[np] - 2.0*sy;
        bz += Sz[nm] + Sz[np] - 2.0*sz;
    }
    
    if(Ny>1){
        nm = indexPBC(i, j-1, k, Nx, Ny, Nz);
        np = indexPBC(i, j+1, k, Nx, Ny, Nz);

        bx += Sx[nm] + Sx[np] - 2.0*sx;
        by += Sy[nm] + Sy[np] - 2.0*sy;
        bz += Sz[nm] + Sz[np] - 2.0*sz;
    }
    
    if(Nz>1){
        nm = indexPBC(i, j, k-1, Nx, Ny, Nz);
        np = indexPBC(i, j, k+1, Nx, Ny, Nz);

        bx += Sx[nm] + Sx[np] - 2.0*sx;
        by += Sy[nm] + Sy[np] - 2.0*sy;
        bz += Sz[nm] + Sz[np] - 2.0*sz;
    }
    

    Bx[I] += bx*J1; By[I] += by*J1; Bz[I] += bz*J1;

    //Shell 2 (12 neighbours)
    bx = 0.0; by = 0.0; bz = 0.0;
    if(Nx>1 && Ny>1){
        nnm1 = indexPBC(i-1, j-1, k, Nx, Ny, Nz);
        nnp1 = indexPBC(i-1, j+1, k, Nx, Ny, Nz);
        nnm2 = indexPBC(i+1, j-1, k, Nx, Ny, Nz);
        nnp2 = indexPBC(i+1, j+1, k, Nx, Ny, Nz);
        
        bx += Sx[nnm1] + Sx[nnp1] + Sx[nnm2] + Sx[nnp2] - 4.0*sx;
        by += Sy[nnm1] + Sy[nnp1] + Sy[nnm2] + Sy[nnp2] - 4.0*sy;
        bz += Sz[nnm1] + Sz[nnp1] + Sz[nnm2] + Sz[nnp2] - 4.0*sz;            
    }
    if(Nx>1 && Nz>1){
        nnm1 = indexPBC(i-1, j, k-1, Nx, Ny, Nz);
        nnp1 = indexPBC(i-1, j, k+1, Nx, Ny, Nz);
        nnm2 = indexPBC(i+1, j, k-1, Nx, Ny, Nz);
        nnp2 = indexPBC(i+1, j, k+1, Nx, Ny, Nz);
        
        bx += Sx[nnm1] + Sx[nnp1] + Sx[nnm2] + Sx[nnp2] - 4.0*sx;
        by += Sy[nnm1] + Sy[nnp1] + Sy[nnm2] + Sy[nnp2] - 4.0*sy;
        bz += Sz[nnm1] + Sz[nnp1] + Sz[nnm2] + Sz[nnp2] - 4.0*sz;            
    }
    if(Ny>1 && Nz>1){
        nnm1 = indexPBC(i, j-1, k-1, Nx, Ny, Nz);
        nnp1 = indexPBC(i, j-1, k+1, Nx, Ny, Nz);
        nnm2 = indexPBC(i, j+1, k-1, Nx, Ny, Nz);
        nnp2 = indexPBC(i, j+1, k+1, Nx, Ny, Nz);
        
        bx += Sx[nnm1] + Sx[nnp1] + Sx[nnm2] + Sx[nnp2] - 4.0*sx;
        by += Sy[nnm1] + Sy[nnp1] + Sy[nnm2] + Sy[nnp2] - 4.0*sy;
        bz += Sz[nnm1] + Sz[nnp1] + Sz[nnm2] + Sz[nnp2] - 4.0*sz;            
    }
    Bx[I] += bx*J2; By[I] += by*J2; Bz[I] += bz*J2;

    //Shell 3 (8 neighbours)
    bx = 0.0; by = 0.0; bz = 0.0;
    if(Nx>1 && Ny>1 && Ny>1){
        nnm1 = indexPBC(i-1, j-1, k-1, Nx, Ny, Nz);
        nnp1 = indexPBC(i-1, j+1, k-1, Nx, Ny, Nz);
        nnm2 = indexPBC(i+1, j-1, k-1, Nx, Ny, Nz);
        nnp2 = indexPBC(i+1, j+1, k-1, Nx, Ny, Nz);

        bx += Sx[nnm1] + Sx[nnp1] + Sx[nnm2] + Sx[nnp2] - 4.0*sx;
        by += Sy[nnm1] + Sy[nnp1] + Sy[nnm2] + Sy[nnp2] - 4.0*sy;
        bz += Sz[nnm1] + Sz[nnp1] + Sz[nnm2] + Sz[nnp2] - 4.0*sz;
        
        nnm1 = indexPBC(i-1, j-1, k+1, Nx, Ny, Nz);
        nnp1 = indexPBC(i-1, j+1, k+1, Nx, Ny, Nz);
        nnm2 = indexPBC(i+1, j-1, k+1, Nx, Ny, Nz);
        nnp2 = indexPBC(i+1, j+1, k+1, Nx, Ny, Nz);
        
        bx += Sx[nnm1] + Sx[nnp1] + Sx[nnm2] + Sx[nnp2] - 4.0*sx;
        by += Sy[nnm1] + Sy[nnp1] + Sy[nnm2] + Sy[nnp2] - 4.0*sy;
        bz += Sz[nnm1] + Sz[nnp1] + Sz[nnm2] + Sz[nnp2] - 4.0*sz;
    }
    Bx[I] += bx*J3; By[I] += by*J3; Bz[I] += bz*J3;

    //Shell 4 (6 neighbours)
    bx = 0.0; by = 0.0; bz = 0.0;
    if(Nx>2){
        nm = indexPBC(i-2, j, k, Nx, Ny, Nz);
        np = indexPBC(i+2, j, k, Nx, Ny, Nz);

        bx += Sx[nm] + Sx[np] - 2.0*sx;
        by += Sy[nm] + Sy[np] - 2.0*sy;
        bz += Sz[nm] + Sz[np] - 2.0*sz;
    }
    
    if(Ny>2){
        nm = indexPBC(i, j-2, k, Nx, Ny, Nz);
        np = indexPBC(i, j+2, k, Nx, Ny, Nz);

        bx += Sx[nm] + Sx[np] - 2.0*sx;
        by += Sy[nm] + Sy[np] - 2.0*sy;
        bz += Sz[nm] + Sz[np] - 2.0*sz;
    }
    
    if(Nz>2){
        nm = indexPBC(i, j, k-2, Nx, Ny, Nz);
        np = indexPBC(i, j, k+2, Nx, Ny, Nz);

        bx += Sx[nm] + Sx[np] - 2.0*sx;
        by += Sy[nm] + Sy[np] - 2.0*sy;
        bz += Sz[nm] + Sz[np] - 2.0*sz;
    }
    Bx[I] += bx*J4; By[I] += by*J4; Bz[I] += bz*J4;
}
