#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"



// generate u1 u2
extern "C" __global__ void
generate_u1u2(float* __restrict__ mx,  float* __restrict__  my,  float* __restrict__ mz,
         float* __restrict__ u1x,  float* __restrict__  u1y,  float* __restrict__ u1z,
         float* __restrict__ u2x,  float* __restrict__  u2y,  float* __restrict__ u2z,
          int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    // bool pp = true;
    
    if (i < N) {


        if(abs(mz[i])<0.5){
            u1x[i] = -mz[i]*mx[i];
            u1y[i] = -mz[i]*my[i];
            u1z[i] = 1.0-mz[i]*mz[i];
        }else{
            u1x[i] = 1.0-mx[i]*mx[i];
            u1y[i] = -mx[i]*my[i];
            u1z[i] = -mx[i]*mz[i];
        }

        float norm = 1.0/sqrt(u1x[i]*u1x[i] + u1y[i]*u1y[i] + u1z[i]*u1z[i]);
        u1x[i] *= norm; u1y[i] *= norm; u1z[i] *= norm;

        u2x[i] = -my[i]*u1z[i] + mz[i]*u1y[i];
        u2y[i] = -mz[i]*u1x[i] + mx[i]*u1z[i];
        u2z[i] = -mx[i]*u1y[i] + my[i]*u1x[i];


    }
}
