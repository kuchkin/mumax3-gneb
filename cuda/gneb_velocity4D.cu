#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"



// velocity rotate
extern "C" __global__ void
velocity4D(float* __restrict__ vx,  float* __restrict__  vy,  float* __restrict__ vz, float* __restrict__ vw,
        float* __restrict__ kx,  float* __restrict__  ky,  float* __restrict__ kz, float* __restrict__ kw,
         float* __restrict__ mx,  float* __restrict__  my,  float* __restrict__ mz, float* __restrict__ mw,
         float* __restrict__ m0x,  float* __restrict__  m0y,  float* __restrict__ m0z, float* __restrict__ m0w,
         int N, int Nz) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    
    if (i < N) {

        float3 m = {mx[i], my[i], mz[i]};
        float3 m0 = {m0x[i], m0y[i], m0z[i]};
        float3 k = {kx[i], ky[i], kz[i]};


        float SIN = dot(m,k)  + mw[i]*kw[i];
        float COS = dot(m,m0) + mw[i]*m0w[i];
        
        // k = k*COS;
        // k -= m0*SIN;

        vx[i] = kx[i]*COS-m0x[i]*SIN;
        vy[i] = ky[i]*COS-m0y[i]*SIN;
        vz[i] = kz[i]*COS-m0z[i]*SIN;
        vw[i] = kw[i]*COS-m0w[i]*SIN;
        

        // if((vx[i]*vx[i]+vy[i]*vy[i]+vz[i]*vz[i]+vw[i]*vw[i])==1){
        //     vx[i] = 100.0;
        //     vy[i] = 100.0;
        //     vz[i] = 100.0;
        //     vw[i] = 100.0;
        // }
    }
}