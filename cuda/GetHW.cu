#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"



// generate u1 u2
extern "C" __global__ void
get_hw(float* __restrict__ k0x,  float* __restrict__  k0y,  float* __restrict__ k0z,
    float* __restrict__ kx,  float* __restrict__  ky,  float* __restrict__ kz,
         float* __restrict__ u1x,  float* __restrict__  u1y,  float* __restrict__ u1z,
         float* __restrict__ u2x,  float* __restrict__  u2y,  float* __restrict__ u2z,
          float* __restrict__ hwx,  float* __restrict__  hwy,int N, float epsilon) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    // bool pp = true;
    
    if (i < N) {


       float Hx = (-k0x[i]+kx[i])/epsilon;
       float Hy = (-k0y[i]+ky[i])/epsilon;
       float Hz = (-k0z[i]+kz[i])/epsilon;

       hwx[i] = u1x[i]*Hx + u1y[i]*Hy + u1z[i]*Hz;
       hwy[i] = u2x[i]*Hx + u2y[i]*Hy + u2z[i]*Hz;
    }
}
