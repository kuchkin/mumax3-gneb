#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"



// velocity rotate
extern "C" __global__ void
velocity_rodrig(float* __restrict__ vx,  float* __restrict__  vy,  float* __restrict__ vz,
    float* __restrict__ kx,  float* __restrict__  ky,  float* __restrict__ kz,
         float* __restrict__ mx,  float* __restrict__  my,  float* __restrict__ mz,
         float* __restrict__ m0x,  float* __restrict__  m0y,  float* __restrict__ m0z,
         int N, int Nz, float dt) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    
    if (i < N) {

        float3 m = {mx[i], my[i], mz[i]};
        float3 m0 = {m0x[i], m0y[i], m0z[i]};
        float3 k = {kx[i], ky[i], kz[i]};

        float3 ds = k*dt;
        float theta = len(ds);
        float3 w = cross(m, ds);
        float p1 = dot(ds,k);
        float p2 = dot(w,k);
        float pref1, pref2;
        if(theta>0.005){
            pref1 = 1.0-theta*theta*(1.0-theta*theta/20.0)/6.0;
            pref2 = 0.5-theta*theta*(1.0-theta*theta/30.0)/24.0;
        }else{
            pref1 = sin(theta)/theta;
            pref2 = (1.0-cos(theta))/(theta*theta);
        }

        

        vx[i] = k.x*cos(theta)-p1*mx[i]*pref1 + w.x*p2*pref2;
        vy[i] = k.y*cos(theta)-p1*my[i]*pref1 + w.y*p2*pref2;
        vz[i] = k.z*cos(theta)-p1*mz[i]*pref1 + w.z*p2*pref2;
        
        // if(abs(vx[i] -1.0)<1.0e-5) vx[i] -= 1.0;
        // if(abs(vy[i] -1.0)<1.0e-5) vy[i] -= 1.0;
        // if(abs(vz[i] -1.0)<1.0e-5) vz[i] -= 1.0;


    }
}
