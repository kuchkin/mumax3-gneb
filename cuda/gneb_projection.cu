#include "hip/hip_runtime.h"

#include "float3.h"

// dst += prefactor * dot(a,b)
extern "C" __global__ void
projection(float* __restrict__ kx, float* __restrict__ ky, float* __restrict__ kz,
           float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
           int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {
        float km = kx[i]*mx[i]+ky[i]*my[i]+kz[i]*mz[i];
        kx[i] = kx[i] - km*mx[i];
        ky[i] = ky[i] - km*my[i];
        kz[i] = kz[i] - km*mz[i];
    }
}

