#include "hip/hip_runtime.h"
#include <stdint.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"
#include "amul.h"

#define MAX(i, j) ( (i>j)? (i)  :  (j) )
#define MIN(i, j) ( (i<j)? (i)  :  (j) )
#define ABS(a) ( (a>0.0)? (a)  :  (-1.0*a) )

// See gneb.go for more details.




extern "C" __global__ void
rgneb(float* __restrict__ Bx, float* __restrict__ By, float* __restrict__ Bz, float* __restrict__ Bw,
    float* __restrict__ Tx, float* __restrict__ Ty, float* __restrict__ Tz, float* __restrict__ Tw,
            float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz, float* __restrict__ mw,
            int Nx, int Ny, int Nz,int noi,int image, float Tp,
            float Lp,float Ln,float k,int CIGNEB, int Pos){

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;
    

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }
    if(iz/(Nz/noi) != image){
        return;
    }

 
    int I = idx(ix, iy, iz);

    float3 T  = make_float3(Tx[I]/sqrtf(Tp),Ty[I]/sqrtf(Tp),Tz[I]/sqrtf(Tp));
    float T4 = Tw[I]/sqrtf(Tp);
    float3 m0 = make_float3(mx[I], my[I], mz[I]);
    float n0 = mw[I];
    float3 B  = make_float3(Bx[I], By[I], Bz[I]);
    float B0 = Bw[I];
    
    float pref;
    if(CIGNEB == 1 && iz/(Nz/noi) == Pos){
        pref = -2.*(dot(B,T)+B0*T4);
    }else{
        pref = k*(Ln-Lp) + dot(B,T)+B0*T4;
    }
    
    B.x = B.x + pref*T.x;
    B.y = B.y + pref*T.y;
    B.z = B.z + pref*T.z;
    B0  = B0  + pref*T4;
    
    
    
    // The perpendicular component of the energy gradient
    // B = B - dot(B,T)*T;
    // T = k*(sqrtf(Lp)-sqrtf(Ln))*T + B;
    // T = k*(Lp-Ln)*T + B;
    pref = dot(m0,B) + n0*B0;
    Bx[I] = B.x - pref*m0.x;
    By[I] = B.y - pref*m0.y;
    Bz[I] = B.z - pref*m0.z;
    Bw[I] = B0  - pref*n0;
    
}


