#include "hip/hip_runtime.h"

#include "float3.h"

// dst += prefactor * dot(a,b)
extern "C" __global__ void
dotproduct1(float* __restrict__ dst, float prefactor,
           float* __restrict__ ax, float* __restrict__ bx, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {
        // dst[i] = 0.0;
        dst[i] += prefactor * ax[i] * bx[i];
    }
}