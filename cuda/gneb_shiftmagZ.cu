#include "hip/hip_runtime.h"
#include "stencil.h"

// shift dst by shy cells (positive or negative) along Z-axis.
// new edge value is clampL at left edge or clampR at right edge.
extern "C" __global__ void
shiftmagz(float* __restrict__  dstX,float* __restrict__  dstY,float* __restrict__  dstZ,
          float* __restrict__  srcX, float* __restrict__  srcY, float* __restrict__  srcZ,
          int Nx,  int Ny,  int Nz, int shz, float clampL, float clampR) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if(ix < Nx && iy < Ny && iz < Nz) {
        int iz2 = iz-shz;
        float3 newval;
        if (iz2 < 0) {
            newval.x = clampL;
            newval.y = clampL;
            newval.z = clampL;
        } else if (iz2 >= Nz) {
            newval.x = clampR;
            newval.y = clampR;
            newval.z = clampR;

        } else {
            newval.x = srcX[idx(ix, iy, iz2)];
            newval.y = srcY[idx(ix, iy, iz2)];
            newval.z = srcZ[idx(ix, iy, iz2)];
        }
        dstX[idx(ix, iy, iz)] = newval.x;
        dstY[idx(ix, iy, iz)] = newval.y;
        dstZ[idx(ix, iy, iz)] = newval.z;
    }
}

