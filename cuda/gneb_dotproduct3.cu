#include "hip/hip_runtime.h"

#include "float3.h"

// dst += prefactor * dot(a,b)
extern "C" __global__ void
dotproduct3(float* __restrict__ dst, float prefactor,
            float* __restrict__ ax, float* __restrict__ ay, float* __restrict__ az, float* __restrict__ aw,
            float* __restrict__ bx, float* __restrict__ by, float* __restrict__ bz, float* __restrict__ bw, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {
        // dst[i] = 0.0;
        float ab = ax[i]*bx[i] + ay[i]*by[i] + az[i]*bz[i] + aw[i]*bw[i];
        float tx = bx[i] - ab*ax[i];
        float ty = by[i] - ab*ay[i];
        float tz = bz[i] - ab*az[i];
        float tw = bw[i] - ab*aw[i];

        dst[i] = prefactor * (tx*tx+ty*ty+tz*tz+tw*tw);
    }
}