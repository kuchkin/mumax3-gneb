#include "hip/hip_runtime.h"

#include "float3.h"

// dst += prefactor * dot(a,b)
extern "C" __global__ void
copy_to_subspace(float* __restrict__ v0x, float* __restrict__ v0y, float* __restrict__ v1x,
           float* __restrict__ v1y, float* __restrict__ w2x, float* __restrict__ w2y,
           float* __restrict__ hwx, float* __restrict__ hwy,
           int N, int id, float alpha, float beta) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {

        

        v1x[i] = w2x[i];
        v1y[i] = w2y[i];

        w2x[i] = hwx[i] - alpha*v1x[i];
        w2y[i] = hwy[i] - alpha*v1y[i];

        if(id>0){
            w2x[i] -= beta*v0x[i];
            w2y[i] -= beta*v0y[i];
        }
        v0x[i] = v1x[i];
        v0y[i] = v1y[i];
    }
}

