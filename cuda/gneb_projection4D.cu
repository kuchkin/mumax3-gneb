#include "hip/hip_runtime.h"
#include "float3.h"

// dst += prefactor * dot(a,b)
extern "C" __global__ void
projection4D(float* __restrict__ kx, float* __restrict__ ky, float* __restrict__ kz, float* __restrict__ kw,
             float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz, float* __restrict__ mw,
           int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {
        float km = kx[i]*mx[i]+ky[i]*my[i]+kz[i]*mz[i]+kw[i]*mw[i];
        kx[i] = kx[i] - km*mx[i];
        ky[i] = ky[i] - km*my[i];
        kz[i] = kz[i] - km*mz[i];
        kw[i] = kw[i] - km*mw[i];
    }
}