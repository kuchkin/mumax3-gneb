#include "hip/hip_runtime.h"
#include "reduce.h"
#include "atomicf.h"
#include "float3.h"

#define proj(mz) ((mz>0.0)? (1.0) : (-1.0))
#define load_vecnorm2(i) (fabs(Bx[i]*(y[i]*y[i] + proj(z[i])*z[i]*(1.0 + proj(z[i])*z[i])) - By[i]*x[i]*y[i] - Bz[i]*x[i]*(proj(z[i]) + z[i]))+fabs(-Bx[i]*x[i]*y[i] + By[i]*(x[i]*x[i] + proj(z[i])*z[i]*(1.0 + proj(z[i])*z[i])) - Bz[i]*y[i]*(proj(z[i]) + z[i])))

extern "C" __global__ void
reducemaxvecnorm2gg(float* __restrict__ x, float* __restrict__ y, float* __restrict__ z, 
					float* __restrict__ Bx, float* __restrict__ By, float* __restrict__ Bz,
 					float* __restrict__ dst, float initVal, int n) {
    reduce(load_vecnorm2, fmax, atomicFmaxabs)
}


/*

// #define load_vecnorm2(i) \
// 	(Bx[i]*x[i]) + (By[i]*y[i]) +  (Bz[i]*z[i])
*/

///torque
///(sqrtf((By[i]*z[i]-Bz[i]*y[i])*(By[i]*z[i]-Bz[i]*y[i]) + (Bx[i]*z[i]-Bz[i]*x[i])*(Bx[i]*z[i]-Bz[i]*x[i]) +  (By[i]*x[i]-Bx[i]*y[i])*(By[i]*x[i]-Bx[i]*y[i])))
