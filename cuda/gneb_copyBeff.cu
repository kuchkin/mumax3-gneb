#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"


extern "C" __global__ void
copyBeff(float* __restrict__  Bx2,  float* __restrict__  By2,  float* __restrict__  Bz2,
        float* __restrict__  Bx,  float* __restrict__  By,  float* __restrict__  Bz,
        int noi, int image, int Nx, int Ny, int Nz){

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if(ix>= Nx || iy>= Ny || iz>=Nz) {
        return;
    }
    int I = (iz*Ny + iy)*Nx + ix;
    float pref = 1.0;
    // if(iz%3!=0) pref = 0.0;
    iz = iz + image*Nz;
    int II = (iz*Ny + iy)*Nx + ix;
    Bx[II] = pref*Bx2[I];
    By[II] = pref*By2[I];
    Bz[II] = pref*Bz2[I];

    // if(iz/(Nz/noi)==image){
    //     Mx2[I] = Mx[((iz%(Nz/noi))*Ny + iy)*Nx + ix];
    //     My2[I] = My[((iz%(Nz/noi))*Ny + iy)*Nx + ix];
    //     Mz2[I] = Mz[((iz%(Nz/noi))*Ny + iy)*Nx + ix];
    // }

}

