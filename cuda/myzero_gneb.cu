#include "hip/hip_runtime.h"

#include "float3.h"


// dst += prefactor * dot(a,b)
extern "C" __global__ void
myzero(float* __restrict__ ax, float* __restrict__ ay, float* __restrict__ az,
           int Nx, int Ny, int Nz) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < Nx*Ny*Nz) {

        if((i/(Nx*Ny))%3!=0) {
           
            ax[i] = 0.0;
            ay[i] = 0.0;
            az[i] = 0.0;
        }
    }
}

