
#include <hip/hip_runtime.h>



extern "C" __global__ void
copyImage(float* __restrict__  Mx,  float* __restrict__  My,  float* __restrict__  Mz,
        float* __restrict__  Mx2,  float* __restrict__  My2,  float* __restrict__  Mz2,
        int noi, int image, int Nx, int Ny, int Nz){

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if(ix>= Nx || iy>= Ny || iz>=Nz) {
        return;
    }
    int I = (iz*Ny + iy)*Nx + ix;
    float pref = 1.0;
    // if(iz%3!=0) pref = 0.0;
    iz = iz + image*Nz;
    int II = (iz*Ny + iy)*Nx + ix;
    Mx2[I] = pref*Mx[II];
    My2[I] = pref*My[II];
    Mz2[I] = pref*Mz[II];
    // vol2[I] = vol[II];
    
    // Mx2[I] = 0.0;
    // My2[I] = 0.0;
    // Mz2[I] = 0.0;
    // if(iz/(Nz/noi)==image){
    //     int I = (iz*Ny + iy)*Nx + ix;
    //     Mx2[((iz%(Nz/noi))*Ny + iy)*Nx + ix] = Mx[I];
    //     My2[((iz%(Nz/noi))*Ny + iy)*Nx + ix] = My[I];
    //     Mz2[((iz%(Nz/noi))*Ny + iy)*Nx + ix] = Mz[I];
    // // }else{
    // //     Mx2[I] = 0.0;
    // //     My2[I] = 0.0;
    // //     Mz2[I] = 0.0;
    // }

}

