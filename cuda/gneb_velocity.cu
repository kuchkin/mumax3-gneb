#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"



// velocity rotate
extern "C" __global__ void
velocity(float* __restrict__ vx,  float* __restrict__  vy,  float* __restrict__ vz,
        float* __restrict__ kx,  float* __restrict__  ky,  float* __restrict__ kz,
         float* __restrict__ mx,  float* __restrict__  my,  float* __restrict__ mz,
         float* __restrict__ m0x,  float* __restrict__  m0y,  float* __restrict__ m0z,
         int N, int Nz) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    
    if (i < N) {

        float3 m = {mx[i], my[i], mz[i]};
        float3 m0 = {m0x[i], m0y[i], m0z[i]};
        float3 k = {kx[i], ky[i], kz[i]};


        float SIN = dot(m,k);
        float COS = dot(m,m0);
        
        // k = k*COS;
        // k -= m0*SIN;

        vx[i] = kx[i]*COS-m0x[i]*SIN;
        vy[i] = ky[i]*COS-m0y[i]*SIN;
        vz[i] = kz[i]*COS-m0z[i]*SIN;
        

        if((vx[i]*vx[i]+vy[i]*vy[i]+vz[i]*vz[i])==1){
            vx[i] = 100.0;
            vy[i] = 100.0;
            vz[i] = 100.0;
        }
    }
}
